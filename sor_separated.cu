#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include "hip/hip_runtime.h"
#include ""
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sequence.h>

#include "util.h"


#define BLOCK_SIZE 16


__global__ static void redKernel(double* rnew, double* r, double* b, int n, int m, double lambda) {
	const int halfn = (n - 1) / 2 + 1;
	const int is_odd = n & 1;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int realj = j * 2 + (i & 1);
	if (i > 0 && i < m - 1 && j + (i & 1) > 0 && realj< n - 1 && (!is_odd || realj < n - 2 || (i & 1))) {
		int left = i * halfn + j - 1 + (i & 1);
		int right = i * halfn + j + (i & 1);
		int top = (i - 1) * halfn + j;
		int bottom = (i + 1) * halfn + j;
		rnew[i * halfn + j] = (1 - lambda) * r[i * halfn + j] + lambda * 0.25 * (b[left] + b[right] 
			+ b[top] + b[bottom]);
	}
}

__global__ static void blackKernel(double* bnew, double* b, double* r, int n, int m, double lambda) {
	const int halfn = (n - 1) / 2 + 1;
	const int is_odd = n & 1;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int realj = j * 2 + ((i + 1) & 1);
	if (i > 0 && i < m - 1 && j + ((i + 1) & 1) > 0 && realj < n-1  && (!is_odd || realj < n - 2 || ((i + 1) & 1))) {
		int left = i * halfn + j - (i & 1);
		int right = i * halfn + j + 1 - (i & 1);
		int top = (i - 1) * halfn + j;
		int bottom = (i + 1) * halfn + j;
		bnew[i * halfn + j] = (1 - lambda) * b[i * halfn + j] + lambda * 0.25 * (r[left] + r[right] 
			+ r[top] + r[bottom]);
	}
}

double calculate_error(thrust::device_vector<double> olddata, thrust::device_vector<double> newdata) {
	auto begin = thrust::make_zip_iterator(thrust::make_tuple(olddata.begin(), newdata.begin()));
	auto end = thrust::make_zip_iterator(thrust::make_tuple(olddata.end(), newdata.end()));
	return thrust::transform_reduce(begin, end, abs_difference(), 0.0, thrust::maximum<double>());
}


extern void sor_separated()
{
	const int n = 13;
	const int m = n;
	int size = n * m;

	const int halfn = (n - 1) / 2 + 1;
	const int is_odd = n & 1;


	thrust::host_vector<double> u(size);

	thrust::fill(u.begin(), u.end(), 0.0);
	thrust::fill(u.begin(), u.begin() + n, 2.0);
	thrust::fill(u.begin() + n * (m - 1), u.end(), 1.0);

	thrust::host_vector<double> h_red(halfn * m);
	thrust::host_vector<double> h_black(halfn * m);

	for (int i = 0; i < m; i++) {
		for (int j = 0; j < halfn; j++) {
			if (2 * j + (i & 1) < n)
				h_red[i * halfn + j] = u[i * n + 2 * j + (i & 1)];
		}
	}

	for (int i = 0; i < m; i++) {
		for (int j = 0; j < halfn; j++) {
			if (2 * j + ((i + 1) & 1) < n)
				h_black[i * halfn + j] = u[i * n + 2 * j + ((i + 1) & 1)];
		}
	}

	thrust::device_vector<double> b(h_black);
	thrust::device_vector<double> bnew(b);
	thrust::device_vector<double> r(h_red);
	thrust::device_vector<double> rnew(r);


	double tol = 1e-5;
	dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
	dim3 gridDim((m - 1) / blockDim.x + 1, (halfn - 1) / blockDim.y + 1);

	double lambda = 1.7;


	double error = tol + 1.0;
	int iterations = 0;
	while (error > tol) {
		iterations++;
		redKernel << <gridDim, blockDim >> > (thrust::raw_pointer_cast(rnew.data()), thrust::raw_pointer_cast(r.data()), thrust::raw_pointer_cast(b.data()), n, m, lambda);
		blackKernel << <gridDim, blockDim >> > (thrust::raw_pointer_cast(bnew.data()), thrust::raw_pointer_cast(b.data()), thrust::raw_pointer_cast(rnew.data()), n, m, lambda);

		error = fmax(calculate_error(r, rnew), calculate_error(b, bnew));
		printf("error is %f\n", error);
		swap(r, rnew);
		swap(b, bnew);
	}

	thrust::host_vector<double> result_red(r);
	thrust::host_vector<double> result_black(b);
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			if ((i + j & 1) == 0) {
				u[i * n + j] = result_red[i * halfn + j / 2];
			}
			else {
				u[i * n + j] = result_black[i * halfn + j / 2];
			}
		}
	}
	print2DArray(thrust::raw_pointer_cast(u.data()), n, m);
	printf("Finished SOR with separation. lambda=%f\n", lambda);
	printf("total iterations: %d\n", iterations);
}

