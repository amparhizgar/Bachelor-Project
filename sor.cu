#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include "hip/hip_runtime.h"
#include ""
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/zip_iterator.h>
#include "util.h"

__device__ static int indexof(int i, int j, int k, int n, int m, int p) {
	return k * n * m + i * n + j;
}

__global__ static void redKernel(double* u, double* un, int n, int m, int p, double lambda) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k = blockIdx.z * blockDim.z + threadIdx.z;
	int index = k * n * m + i * n + j;

	if ((i + j + k) % 2 == 0) {
		if (i > 0 && i < m - 1 && j > 0 && j < n - 1 && k > 0 && k < p - 1) {
			un[index] = (1 - lambda) * u[index] + lambda / 6.0 * (u[indexof(i - 1, j, k, n, m, p)] + u[indexof(i + 1, j, k, n, m, p)]
				+ u[indexof(i, j - 1, k, n, m, p)] + u[indexof(i, j + 1, k, n, m, p)]
				+ u[indexof(i, j, k - 1, n, m, p)] + u[indexof(i, j, k + 1, n, m, p)]);
		}
	}
}

__global__ static void blackKernel(double* u, double* un, int n, int m, int p, double lambda) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k = blockIdx.z * blockDim.z + threadIdx.z;
	int index = k * n * m + i * n + j;

	if ((i + j + k) % 2 != 0) {
		if (i > 0 && i < m - 1 && j > 0 && j < n - 1 && k > 0 && k < p - 1) {
			un[index] = (1 - lambda) * u[index] + lambda / 6.0 * (un[indexof(i - 1, j, k, n, m, p)] + un[indexof(i + 1, j, k, n, m, p)]
				+ un[indexof(i, j - 1, k, n, m, p)] + un[indexof(i, j + 1, k, n, m, p)]
				+ un[indexof(i, j, k - 1, n, m, p)] + un[indexof(i, j, k + 1, n, m, p)]);
		}
	}
}

__global__ static void errorKernel(double* u, double* un, double* error, int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size)
		error[index] = fabs(u[index] - un[index]);
}

extern Result sor(thrust::device_vector<double>& u, int n, int m, int p, ConvergenceCriteria cc)
{
	int size = n * m * p;
	thrust::device_vector<double> un(u);
	thrust::device_vector<double> error_temp(size);

	dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
	dim3 gridDim((m - 1) / blockDim.x + 1, (n - 1) / blockDim.y + 1, (p - 1) / blockDim.z + 1);

	double lambda = 2 / (1 + sin(pi / (n + 1)));

	double error;
	int iterations = 0;
	while (true) {
		iterations++;
		redKernel << <gridDim, blockDim >> > (thrust::raw_pointer_cast(u.data()), thrust::raw_pointer_cast(un.data()), n, m, p, lambda);
		blackKernel << <gridDim, blockDim >> > (thrust::raw_pointer_cast(u.data()), thrust::raw_pointer_cast(un.data()), n, m, p, lambda);

		errorKernel << <(size - 1) / 128 + 1, 128 >> > (thrust::raw_pointer_cast(u.data()), thrust::raw_pointer_cast(un.data()),
			thrust::raw_pointer_cast(error_temp.data()), size);
		checkForError();
		error = thrust::reduce(error_temp.begin(), error_temp.end(), 0.0, thrust::maximum<double>());
		swap(u, un);
		if (cc.hasConverged(error, iterations))
			break;
	}

	return Result(&u, error, iterations);
}

